#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <thread>
#include <chrono>
#include "args.h"     // Include the args parser
#include "parser.h"   // Include the file reader
#include "work.cuh"   // Include the work queue
#include "memory.cuh" // Include the memory manager
#include "mine.cuh"

#define KILO 1024ULL
#define MEGA KILO *KILO
#define GIGA KILO *MEGA

#define page_size (128 * KILO)
#define total_memory (5 * GIGA)


// make && ./cuEFIM '/home/tarun/testing/test.txt' 5 \\s
// make && time ./cuEFIM '/home/tarun/cuEFIM/datasets/accidents_utility_spmf.txt' 15000000 \\s

__global__ void copy_work(AtomicWorkStack *curr_work_queue, WorkItem *work_item, int *primary, int primary_size)
{
    int tid = threadIdx.x;
    __shared__ WorkItem item;
    // WorkItem item;
    if (tid == 0)
    {
        item.pattern = (int *)global_malloc(sizeof(int));
        item.pattern_length = 0;
        item.utility = 0;
        item.db = (Database *)global_malloc(sizeof(Database));
        item.db->numItems = 0;
        item.db->d_data = (Item *)global_malloc(sizeof(Item) * work_item->db->numItems);
        item.db->d_transactions = (Transaction *)global_malloc(sizeof(Transaction) * work_item->db->numTransactions);
        item.db->numTransactions = work_item->db->numTransactions;
        item.max_item = work_item->max_item;

        item.work_done = (int *)global_malloc(sizeof(int));
        item.work_done[0] = 0;
        item.work_count = primary_size;
    }

    __syncthreads();

    for (int i = tid; i < work_item->db->numItems; i += blockDim.x)
    {
        item.db->d_data[i] = work_item->db->d_data[i];
    }

    __syncthreads();

    for (int i = tid; i < work_item->db->numTransactions; i += blockDim.x)
    {
        item.db->d_transactions[i].data = item.db->d_data + (work_item->db->d_transactions[i].data - work_item->db->d_data);
        item.db->d_transactions[i].utility = 0;
        item.db->d_transactions[i].length = work_item->db->d_transactions[i].length;
    }

    __syncthreads();


    if (tid == 0)
    {
        for (int i = 0; i < primary_size; i++)
        {
            item.primary = primary[i];
            curr_work_queue->push(item);
        }
    }
   
}

std::map<std::string, int> parse_patterns(int *d_high_utility_patterns, std::unordered_map<int, std::string> rename)

{
    std::map<std::string, int> Patterns;
    int duplicate = 0;
    int util_dup = 0;

    // convert high utility patterns to string
    std::string high_utility_patterns_str = "";
    std::vector<int> high_utility_patten;
    for (int i = 0; i < d_high_utility_patterns[1]; i++)
    {
        while (d_high_utility_patterns[i + 2] != 0)
        {
            high_utility_patten.push_back(d_high_utility_patterns[i + 2]);
            i++;
        }

        // if empty, skip
        if (high_utility_patten.size() == 0)
        {
            continue;
        }

        for (int j = 0; j < high_utility_patten.size() - 1; j++)
        {
            high_utility_patterns_str += rename[high_utility_patten[j]] + " ";
        }

        //

        // Patterns[high_utility_patterns_str] = high_utility_patten[high_utility_patten.size() - 1];
        if (Patterns.find(high_utility_patterns_str) == Patterns.end())
        {
            Patterns[high_utility_patterns_str] = high_utility_patten[high_utility_patten.size() - 1];
        }
        else
        {
            duplicate++;
            // print old util and new util
            // std::cout << "Old: " << Patterns[high_utility_patterns_str] << " New: " << high_utility_patten[high_utility_patten.size() - 1] << "\n";
            if (Patterns[high_utility_patterns_str] == high_utility_patten[high_utility_patten.size() - 1])
            {
                util_dup++;
            }
        }

        high_utility_patterns_str = "";
        high_utility_patten.clear();
    }
    return Patterns;
}

int main(int argc, char *argv[])
{
    hipError_t cudaStatus;

    int gpu_max_shared_mem;
    hipDeviceGetAttribute(&gpu_max_shared_mem, hipDeviceAttributeSharedMemPerBlockOptin, 0);
    std::cout << "Max Shared Memory: " << gpu_max_shared_mem << " Bytes\n";

    // increase cuda stack size
    // hipDeviceSetLimit(hipLimitStackSize, 32 * 1024);
    // Make CPU not poll
    hipError_t err = hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to set device flags: %s\n", hipGetErrorString(err));
        return -1;
    }

    // increase stack size
    // hipDeviceSetLimit(hipLimitStackSize, 64 * 1024);
    Timer timer;

    // Parse command-line arguments using args_parser
    ParsedArgs args;
    if (!parseArguments(argc, argv, args))
    {
        // Parsing failed; exit the program
        return EXIT_FAILURE;
    }


    timer.recordPoint("Start");
    ReadFileResult fileResult = read_file(args.filename, args.separator, args.utility);
    timer.recordPoint("File Read");

    // Access the parsed data
    auto &filteredTransactions = fileResult.filteredTransactions;
    auto &primary = fileResult.primary;
    auto &rename = fileResult.rename;
    int max_item = fileResult.max_item;

    // Flatten filteredTransactions
    std::vector<Item> items;
    std::vector<int> start;
    std::vector<int> end;

    int max_size = 0;
    for (const auto &[key, val] : filteredTransactions)
    {
        max_size = std::max(max_size, (int)key.size());
        start.push_back(items.size());
        for (int i = 0; i < key.size(); i++)
        {
            items.push_back({key[i], val[i]});
        }
        end.push_back(items.size());
    }
    std::cout << "Largest Transaction: " << max_size << "\n";

    /*
        // Use shared memory only for values that one block will process together.
    __shared__ WorkItem work_item; // the work-item popped from the queue
    __shared__ bool s_popped;      // did we successfully pop a work-item?

    // Shared copies for data that one block uses to process the work-item.
    __shared__ WorkItem new_work_item;
    __shared__ Transaction *temp_transaction;
    __shared__ Item *local_util;
    __shared__ int num_items;
    __shared__ int num_transactions;

    __shared__ int *hashes;
    __shared__ Item *subtree_util;
    __shared__ int max_item;
    __shared__ int primary_count;
    */

    int shared_mem_req = max_item * sizeof(Utils) * scale // for local_util
                        // + max_size * sizeof(Transaction) // for temp_transaction
                        + KILO; // for other variables

    std::cout << "Shared Memory Required: " << shared_mem_req << " Bytes\n";

    if (shared_mem_req > gpu_max_shared_mem) {
        std::cout << "Requested shared memory exceeds GPU max;\n";
        // shared_mem_req = gpu_max_shared_mem;
        return -1;
    }

    int32_t *d_high_utility_patterns;
    hipMallocManaged(&d_high_utility_patterns, 128 * MEGA);
    memset(d_high_utility_patterns, 0, 128 * MEGA);
    d_high_utility_patterns[1] = 2;

    init_global_allocator(total_memory, 0);

    // CudaMemoryManager *mm = createCudaMemoryManager(total_memory, page_size);
    // std::cout << "Memory Manager Initialized\n";

    AtomicWorkStack *curr_work_queue;
    hipMallocManaged(&curr_work_queue, sizeof(AtomicWorkStack));
    curr_work_queue->init();

    WorkItem *work_item;
    hipMallocManaged(&work_item, sizeof(WorkItem));
    work_item->pattern = nullptr;
    work_item->pattern_length = 0;
    work_item->work_count = primary.size();
    work_item->max_item = max_item;
    work_item->work_done = nullptr;

    Database *db;
    hipMallocManaged(&db, sizeof(Database));
    db->numItems = items.size();
    db->d_data = nullptr;
    db->d_transactions = nullptr;
    db->numTransactions = start.size();

    hipMallocManaged(&db->d_data, items.size() * sizeof(Item));
    hipMemcpy(db->d_data, items.data(), items.size() * sizeof(Item), hipMemcpyHostToDevice);

    hipMallocManaged(&db->d_transactions, start.size() * sizeof(Transaction));
    for (int i = 0; i < start.size(); i++)
    {
        db->d_transactions[i].data = db->d_data + start[i];
        db->d_transactions[i].utility = 0;
        db->d_transactions[i].length = end[i] - start[i];
    }

    work_item->db = db;

    int *d_primary;
    hipMallocManaged(&d_primary, primary.size() * sizeof(int));
    hipMemcpy(d_primary, primary.data(), primary.size() * sizeof(int), hipMemcpyHostToDevice);

    copy_work<<<1, threads>>>(curr_work_queue, work_item, d_primary, primary.size());
    hipDeviceSynchronize();
    timer.recordPoint("Data Copy to GPU");

    hipFree(d_primary);
    hipFree(db->d_data);
    hipFree(db->d_transactions);
    hipFree(db);



    while (curr_work_queue->active > 0)
    {
        printf("Top: %d\n", curr_work_queue->active);
        test<<<blocks, threads,shared_mem_req>>>(curr_work_queue, d_high_utility_patterns, args.utility);
        // print last error
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return -1;
        }

        hipDeviceSynchronize();
    }

    timer.recordPoint("Kernel Execution");
    std::map<std::string, int> Patterns = parse_patterns(d_high_utility_patterns, rename);
    // timer
    hipFree(d_high_utility_patterns);

    // for (const auto &p : Patterns)
    // {
    //     std::cout << p.first << "UTIL: " << p.second << std::endl;
    // }

    std::cout << "High Utility Patterns: " << Patterns.size() << "\n";

    // print_global_stats();

    free_global_allocator();

    timer.printRecords();

    return 0;
}