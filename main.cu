#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <thread>
#include "args.h"     // Include the args parser
#include "parser.h"   // Include the file reader
#include "work.cuh"   // Include the work queue
#include "memory.cuh" // Include the memory manager

#define KILO 1024ULL
#define MEGA KILO *KILO
#define GIGA KILO *MEGA

#define scale 2
#define page_size 512
#define total_memory 6 * GIGA

#define blocks 1

__global__ void copy(
    CudaMemoryManager *mm,
    AtomicWorkStack *work_queue,
    Item *items,
    int *start,
    int *end,
    int *primary,
    int num_primary,
    int num_transactions,
    int max_item)
{

    stack_init(work_queue);

    int item_count = end[num_transactions - 1] - start[0];

    WorkItem work_item;

    // pattern
    work_item.pattern = reinterpret_cast<int *>(mm->malloc((2 + num_primary) * sizeof(int)));

    // db
    work_item.db = reinterpret_cast<Database *>(mm->malloc(sizeof(Database)));

    // items
    // work_item.db->d_data = reinterpret_cast<Item *>(base_ptr);
    work_item.db->d_data = reinterpret_cast<Item *>(mm->malloc(item_count * sizeof(Item)));
    memcpy(work_item.db->d_data, items, item_count * sizeof(Item));
    work_item.db->numItems = item_count;

    // transactions
    work_item.db->d_transactions = reinterpret_cast<Transaction *>(mm->malloc(num_transactions * sizeof(Transaction)));
    work_item.db->numTransactions = num_transactions;
    for (int i = 0; i < num_transactions; i++)
    {
        work_item.db->d_transactions[i].data = work_item.db->d_data;
        work_item.db->d_transactions[i].start = start[i];
        work_item.db->d_transactions[i].end = end[i];
    }
    // counts
    work_item.db->numItems = item_count;
    work_item.db->numTransactions = num_transactions;

    work_item.work_done = reinterpret_cast<int *>(mm->malloc(sizeof(int)));
    work_item.work_count = num_primary;
    work_item.max_item = max_item;

    for (int i = 0; i < num_primary; i++)
    {
        work_item.primary = primary[i];

        stack_push(work_queue, work_item);
    }
}

__device__ void printDB(Database *db)
{
    printf("DB: \n");
    for (int i = 0; i < db->numTransactions; i++)
    {
        printf("%d|", db->d_transactions[i].utility);
        for (int j = 0; j < db->d_transactions[i].length(); j++)
        {
            printf("%d:%d ", db->d_transactions[i].get()[j].key, db->d_transactions[i].get()[j].util);
        }
        printf("\n");
    }
    // printf("\n\n");
}


// __device__ void add_local_util(local_util, old->max_item * scale, old->db->d_data[i].key, total_util);
__device__ void add_bucket_util(Item *local_util, int max_item, int key, int total_util)
{
    // hash the key
    int idx = hashFunction(key, max_item * scale);

    // find the key
    while (true)
    {
        // we are adding in atomic so do compare and swap for the key
        int old = atomicCAS(&local_util[idx].key, 0, key);
        if (old == key)
        {
            // if the key is already present, add the utility
            atomicAdd(&local_util[idx].util, total_util);
            return;
        }
        else if (old == 0)
        {
            // if the key is not present, add the key and utility
            atomicExch(&local_util[idx].key, key);
            atomicAdd(&local_util[idx].util, total_util);
            return;
        }
        // if the key is not present, find the next slot
        idx = (idx + 1) % (max_item * scale);
    }


}


__global__ void project(WorkItem *old, WorkItem *curr, Item *local_util)
{

    int tid = blockIdx.x;

    if (tid >= old->db->numTransactions) return;

    // find the item in old
    int item = curr->pattern[curr->pattern[0]];
    int idx = old->db->d_transactions[tid].findItem(item);
    if (idx == -1)
    {
        atomicAdd(&curr->db->transaction_tracker, 1);

        return;
    }

    int items_this_trans = old->db->d_transactions[tid].end - (idx + 1);
    int ret = atomicAdd(&curr->db->numItems, items_this_trans);
    int tran_ret = atomicAdd(&curr->db->numTransactions, 1);

    curr->db->d_transactions[tran_ret].utility = old->db->d_transactions[tid].utility + old->db->d_data[idx].util;
    curr->db->d_transactions[tran_ret].data = curr->db->d_data;

    // write.

    // update the db
    curr->db->d_transactions[tran_ret].start = ret;

    int total_util = curr->db->d_transactions[tran_ret].utility;
    for (int i = idx+1; i < old->db->d_transactions[tid].end; i++)
    {
        total_util += old->db->d_data[i].util;
    }

    for (int i = idx+1; i < old->db->d_transactions[tid].end; i++)
    {
    //     ret++;
        curr->db->d_data[ret++] = old->db->d_data[i];
        add_bucket_util(local_util, old->max_item * scale, old->db->d_data[i].key, total_util);
    }
    curr->db->d_transactions[tran_ret].end = ret;

    atomicAdd(&curr->db->transaction_tracker, 1);
}

__device__ void printBucketUtil(Item *local_util, int max_item)
{
    printf("Bucket Util: \t");
    for (int i = 0; i < max_item; i++)
    {
        // if (local_util[i].key != 0)
        {
            printf("%d:%d ", local_util[i].key, local_util[i].util);
        }
    }
    printf("\n");
}


__global__ void trim_and_merge(WorkItem *curr, Item *local_util, Item *hashes, Item *subtree_util, int min_util)
{
    int tid = blockIdx.x;

    if (tid >= curr->db->numTransactions) return;
    int curr_loc = 0;

    for (int i = 0; i < curr->db->d_transactions[tid].length(); i++)
    {
        int idx = hashFunction(curr->db->d_transactions[tid].get()[i].key, curr->max_item * scale);
        printf("TID:%d\tItem: %d\tLocal Util Idx: %d\tLocal Util: %d\n", tid, curr->db->d_transactions[tid].get()[i].key, idx, local_util[idx].util);
        if (local_util[idx].util >= min_util)
        {
            // make the item be written to curr_loc in the transaction
            curr->db->d_data[curr->db->d_transactions[tid].start + curr_loc] = curr->db->d_transactions[tid].get()[i];
            curr_loc++;
        }


    }

    // update the transaction length
    curr->db->d_transactions[tid].end = curr->db->d_transactions[tid].start + curr_loc;
    printf("Transaction Length: %d\n", curr->db->d_transactions[tid].length());


    atomicAdd(&curr->db->transaction_tracker, 1);
}

__global__ void verify(CudaMemoryManager *mm, AtomicWorkStack *work_queue, int min_util)
{
    WorkItem *work_item = reinterpret_cast<WorkItem *>(mm->malloc(sizeof(WorkItem)));
    printf("Work Count: %d\n", stack_get_work_count(work_queue));

    while (stack_get_work_count(work_queue) > 0)
    {
        stack_pop(work_queue, work_item);

        printf("Pattern: ");
        for (int i = 0; i < work_item->pattern[0]; i++)
        {
            printf("%d ", work_item->pattern[i + 1]);
        }
        printf("(%d)\n", work_item->primary);

        printDB(work_item->db);


        WorkItem *new_work_item = reinterpret_cast<WorkItem *>(mm->malloc(sizeof(WorkItem)));


        // pattern
        new_work_item->pattern = reinterpret_cast<int *>(mm->malloc((work_item->pattern[0] + 2) * sizeof(int)));
        memcpy(new_work_item->pattern, work_item->pattern, (work_item->pattern[0]) * sizeof(int));
        new_work_item->pattern[++new_work_item->pattern[0]] = work_item->primary;
        // base_ptr += (work_item->pattern[0] + 2) * sizeof(int);

        // db
        new_work_item->db = reinterpret_cast<Database *>(mm->malloc(sizeof(Database)));


        new_work_item->db->d_data = reinterpret_cast<Item *>(mm->malloc(work_item->db->numItems * sizeof(Item)));


        new_work_item->db->d_transactions = reinterpret_cast<Transaction *>(mm->malloc(work_item->db->numTransactions * sizeof(Transaction)));


        // Item *local_util = reinterpret_cast<Item *>(deviceMemMalloc(memory_manager, work_item->max_item * scale * sizeof(Item)));
        Item *local_util = reinterpret_cast<Item *>(mm->malloc(work_item->max_item * scale * sizeof(Item)));


        project<<<work_item->db->numTransactions, 1>>>(work_item, new_work_item, local_util);

        while (new_work_item->db->transaction_tracker != work_item->db->numTransactions)
        {
            __threadfence();
        }
        printf("Number of Transactions: %d\n", new_work_item->db->numTransactions);
        printf("Number of Items: %d\n", new_work_item->db->numItems);

        printDB(new_work_item->db);
        printBucketUtil(local_util, work_item->max_item * scale);

        if (new_work_item->db->numTransactions == 0)
        {
            mm->free(new_work_item->pattern);
            mm->free(new_work_item->db->d_data);
            mm->free(new_work_item->db->d_transactions);
            mm->free(new_work_item->db);

            mm->free(local_util);
            atomicSub(&work_queue->active, 1);
            int ret = atomicAdd(&work_item->work_done[0], 1);
            if (ret == work_item->work_count - 1)
            {

                mm->free(work_item->pattern);
                mm->free(work_item->db->d_data);
                mm->free(work_item->db->d_transactions);
                mm->free(work_item->db);
                mm->free(work_item->work_done);

            } 

            continue;
        }

        // trim and merge
        Item *hashes = reinterpret_cast<Item *>(mm->malloc(new_work_item->db->numTransactions * sizeof(Item) * scale));
        Item *subtree_util = reinterpret_cast<Item *>(mm->malloc(work_item->max_item * scale * sizeof(Item)));
        new_work_item->db->transaction_tracker = 0;
        new_work_item->max_item = work_item->max_item;

        trim_and_merge<<<new_work_item->db->numTransactions, 1>>>(new_work_item, local_util, hashes, subtree_util, min_util);

        while (new_work_item->db->transaction_tracker != new_work_item->db->numTransactions)
        {
            __threadfence();
        }

        printDB(new_work_item->db);

  
        // // print new work item
        // printf("Pattern: (len:%d) | ", new_work_item.pattern[0]);
        // for (int i = 0; i < new_work_item.pattern[0]; i++)
        // {
        //     printf("%d ", new_work_item.pattern[i + 1]);
        // }

        // printf("\n");

        // printf("Primary: %d\n", new_work_item.primary);
        // printf("Work Done: %d\n", new_work_item.work_done[0]);
        // printf("Work Count: %d|", new_work_item.work_count);
        // printf("Bytes: %d\n", new_work_item.bytes_to_alloc);
        // printf("DB: \n");
        // for (int i = 0; i < new_work_item.db->numTransactions; i++)
        // {
        //     printf("%d|", new_work_item.db->d_transactions[i].utility);
        //     for (int j = 0; j < new_work_item.db->d_transactions[i].length(); j++)
        //     {
        //         printf("%d:%d ", new_work_item.db->d_transactions[i].get()[j].key, new_work_item.db->d_transactions[i].get()[j].util);
        //     }
        //     printf("\n");
        // }
        // printf("\n");

        // new_work_item.db = reinterpret_cast<Database *>(base_ptr);
        // memcpy(new_work_item.db, work_item.db, sizeof(Database));

        // // Local Util, Subtree Util
        // int bytes_for_util = 2 * work_item->max_item * scale * sizeof(Item);
        // void *n_base_ptr = deviceMemMalloc(memory_manager, bytes_for_util);

        // Item *local_util = reinterpret_cast<Item *>(n_base_ptr);
        // n_base_ptr += bytes_for_util;

        // Item *subtree_util = reinterpret_cast<Item *>(n_base_ptr);
        // n_base_ptr += bytes_for_util;

        // // Tran Hash
        // Item *tran_hash = reinterpret_cast<Item *>(deviceMemMalloc(memory_manager, work_item->db->numTransactions * sizeof(Item) * scale));

        atomicSub(&work_queue->active, 1);
        int ret = atomicAdd(&work_item->work_done[0], 1);
        if (ret == work_item->work_count - 1)
        {
            // deviceMemFree(memory_manager, work_item->base_ptr, work_item->bytes_to_alloc);
            // mm->free(work_item->base_ptr);
            mm->free(work_item->pattern);
            mm->free(work_item->db->d_data);
            mm->free(work_item->db->d_transactions);
            mm->free(work_item->db);
            mm->free(work_item->work_done);
        }
        printf("\n");
        printf("Work Count: %d\n", stack_get_work_count(work_queue));
    }
}

__global__ void mine(CudaMemoryManager *memory_manager, AtomicWorkStack *work_queue, int utility, int32_t *high_utility_patterns)
{
    WorkItem work_item;

    // while
}

int main(int argc, char *argv[])
{
    // Parse command-line arguments using args_parser
    ParsedArgs args;
    if (!parseArguments(argc, argv, args))
    {
        // Parsing failed; exit the program
        return EXIT_FAILURE;
    }

    // increase cuad stack size
    // hipDeviceSetLimit(hipLimitStackSize, 32 * 1024);

    ReadFileResult fileResult = read_file(args.filename, args.separator, args.utility);

    // Access the parsed data
    auto &filteredTransactions = fileResult.filteredTransactions;
    auto &primary = fileResult.primary;
    auto &rename = fileResult.rename;
    int max_item = fileResult.max_item;

    // Flatten filteredTransactions
    std::vector<Item> items;
    std::vector<int> start;
    std::vector<int> end;

    for (const auto &[key, val] : filteredTransactions)
    {
        start.push_back(items.size());
        for (int i = 0; i < key.size(); i++)
        {
            std::cout << key[i] << ":" << val[i] << " ";
            items.push_back({key[i], val[i]});
        }
        std::cout << "\n";
        end.push_back(items.size());
    }

    // copy items to device
    Item *d_items;
    int *d_start;
    int *d_end;
    int *d_primary;

    size_t num_items = items.size();
    size_t num_transactions = start.size();

    hipMalloc(&d_items, num_items * sizeof(Item));
    hipMalloc(&d_start, num_transactions * sizeof(int));
    hipMalloc(&d_end, num_transactions * sizeof(int));
    hipMalloc(&d_primary, primary.size() * sizeof(int));

    hipMemcpy(d_items, items.data(), num_items * sizeof(Item), hipMemcpyHostToDevice);
    hipMemcpy(d_start, start.data(), num_transactions * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_end, end.data(), num_transactions * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_primary, primary.data(), primary.size() * sizeof(int), hipMemcpyHostToDevice);

    int32_t *d_high_utility_patterns;
    hipMallocManaged(&d_high_utility_patterns, 128 * MEGA); // 1GB
    d_high_utility_patterns[1] = 2;

    // memory
    // std::cout << "Allocating Memory: " << page_count * page_size << " bytes\t(MB: " << (page_count * page_size) / (MEGA) << ")\n";

    // CudaMemoryManager *memory_manager = createMemoryManager(total_memory, page_size);
    CudaMemoryManager *mm = createCudaMemoryManager(total_memory / page_size, page_size);

    // start work queue
    AtomicWorkStack *work_queue;
    hipMalloc(&work_queue, sizeof(AtomicWorkStack));

    copy<<<1, 1>>>(mm, work_queue, d_items, d_start, d_end, d_primary, primary.size(), num_transactions, max_item);
    // Synchronize to ensure kernel completion
    hipDeviceSynchronize();
        std::cout << "\n";
        std::cout << "\n";


    verify<<<1, 1>>>(mm, work_queue, args.utility);

    // // free the memory
    // hipFree(d_items);
    // hipFree(d_start);
    // hipFree(d_end);
    // hipFree(d_primary);

    // mine<<<blocks, 1>>>(memory_manager, work_queue, args.utility, d_high_utility_patterns);
    hipDeviceSynchronize();

    std::cout << "High Utility Patterns: " << d_high_utility_patterns[0] << "\n";

    std::map<std::string, int> Patterns;

    // convert high utility patterns to string
    std::string high_utility_patterns_str = "";
    std::vector<int> high_utility_patten;
    for (int i = 0; i < d_high_utility_patterns[1]; i++)
    {
        while (d_high_utility_patterns[i + 2] != 0)
        {
            high_utility_patten.push_back(d_high_utility_patterns[i + 2]);
            i++;
        }
        // if empty, skip
        if (high_utility_patten.size() == 0)
        {
            continue;
        }

        for (int j = 0; j < high_utility_patten.size() - 1; j++)
        {
            high_utility_patterns_str += rename[high_utility_patten[j]] + " ";
        }
        Patterns[high_utility_patterns_str] = high_utility_patten[high_utility_patten.size() - 1];

        high_utility_patterns_str = "";
        high_utility_patten.clear();
    }
    for (const auto &p : Patterns)
    {
        std::cout << p.first << "UTIL: " << p.second << std::endl;
    }

    std::cout << "High Utility Patterns: " << d_high_utility_patterns[0] << "\n";

    return 0;
}