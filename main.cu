#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <thread>
#include <chrono>
#include "args.h"     // Include the args parser
#include "parser.h"   // Include the file reader
#include "work.cuh"   // Include the work queue
#include "memory.cuh" // Include the memory manager
#include "mine.cuh"

#define KILO 1024ULL
#define MEGA KILO *KILO
#define GIGA KILO *MEGA

#define page_size (128 * KILO)
#define total_memory (6 * GIGA)

// make && ./cuEFIM '/home/tarun/testing/test.txt' 5 \\s
// make && time ./cuEFIM '/home/tarun/cuEFIM/datasets/accidents_utility_spmf.txt' 15000000 \\s
__global__ void copy_work(AtomicWorkStack *curr_work_queue, WorkItem *work_item, int *primary, int primary_size)
{
    int tid = threadIdx.x;
    __shared__ WorkItem item;
    // WorkItem item;
    if (tid == 0)
    {
        item.pattern = (int *)global_malloc(sizeof(int));
        item.pattern_length = 0;
        item.utility = 0;
        item.db = (Database *)global_malloc(sizeof(Database));
        item.db->numItems = 0;
        item.db->d_data = (Item *)global_malloc(sizeof(Item) * work_item->db->numItems);
        item.db->d_transactions = (Transaction *)global_malloc(sizeof(Transaction) * work_item->db->numTransactions);
        item.db->numTransactions = work_item->db->numTransactions;
        item.max_item = work_item->max_item;

        item.work_done = (int *)global_malloc(sizeof(int));
        item.work_done[0] = 0;
        item.work_count = primary_size;
    }

    __syncthreads();

    for (int i = tid; i < work_item->db->numItems; i += blockDim.x)
    {
        item.db->d_data[i] = work_item->db->d_data[i];
    }

    __syncthreads();

    for (int i = tid; i < work_item->db->numTransactions; i += blockDim.x)
    {
        item.db->d_transactions[i].data = item.db->d_data + (work_item->db->d_transactions[i].data - work_item->db->d_data);
        item.db->d_transactions[i].utility = 0;
        item.db->d_transactions[i].length = work_item->db->d_transactions[i].length;
    }

    __syncthreads();

    if (tid == 0)
    {
        for (int i = 0; i < primary_size; i++)
        {
            item.primary = primary[i];
            curr_work_queue->push(item);
        }
    }
}

std::map<std::string, int> parse_patterns(int *d_high_utility_patterns, std::unordered_map<int, std::string> rename)

{
    std::map<std::string, int> Patterns;
    int duplicate = 0;
    int util_dup = 0;

    // convert high utility patterns to string
    std::string high_utility_patterns_str = "";
    std::vector<int> high_utility_patten;
    for (int i = 0; i < d_high_utility_patterns[1]; i++)
    {
        while (d_high_utility_patterns[i + 2] != 0)
        {
            high_utility_patten.push_back(d_high_utility_patterns[i + 2]);
            i++;
        }

        // if empty, skip
        if (high_utility_patten.size() == 0)
        {
            continue;
        }

        for (int j = 0; j < high_utility_patten.size() - 1; j++)
        {
            high_utility_patterns_str += rename[high_utility_patten[j]] + " ";
        }

        Patterns[high_utility_patterns_str] = high_utility_patten[high_utility_patten.size() - 1];

        high_utility_patterns_str = "";
        high_utility_patten.clear();
    }
    return Patterns;
}

int main(int argc, char *argv[])
{
    hipError_t cudaErr;

    hipDeviceProp_t deviceProp;
    cudaErr = hipGetDeviceProperties(&deviceProp, 0);
    if (cudaErr != hipSuccess)
    {
        std::cerr << "Error: " << hipGetErrorString(cudaErr) << std::endl;
        return 1;
    }

    // Calculate the theoretical max concurrent threads
    int maxConcurrentThreads = deviceProp.multiProcessorCount * deviceProp.maxThreadsPerMultiProcessor;

    std::cout << "Device " << 0 << ": " << deviceProp.name << std::endl;
    std::cout << "Number of SMs: " << deviceProp.multiProcessorCount << std::endl;
    std::cout << "Max threads per SM: " << deviceProp.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "Theoretical max concurrent threads: "
              << maxConcurrentThreads << std::endl;

    int block_count = maxConcurrentThreads / deviceProp.maxThreadsPerBlock;
    std::cout << "Block count: " << maxConcurrentThreads / threads << std::endl;

    // increase cuda stack size
    // hipDeviceSetLimit(hipLimitStackSize, 32 * 1024);
    // Make CPU not poll
    cudaErr = hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
    if (cudaErr != hipSuccess)
    {
        fprintf(stderr, "Failed to set device flags: %s\n", hipGetErrorString(cudaErr));
        return -1;
    }

    // increase stack size
    // hipDeviceSetLimit(hipLimitStackSize, 64 * 1024);
    Timer timer;

    // Parse command-line arguments using args_parser
    ParsedArgs args;
    if (!parseArguments(argc, argv, args))
    {
        // Parsing failed; exit the program
        return EXIT_FAILURE;
    }

    timer.recordPoint("Start");
    ReadFileResult fileResult = read_file(args.filename, args.separator, args.utility);
    timer.recordPoint("File Read");

    // Access the parsed data
    auto &filteredTransactions = fileResult.filteredTransactions;
    auto &primary = fileResult.primary;
    auto &rename = fileResult.rename;
    int max_item = fileResult.max_item;

    // Flatten filteredTransactions
    std::vector<Item> items;
    std::vector<int> start;
    std::vector<int> end;

    for (const auto &[key, val] : filteredTransactions)
    {
        start.push_back(items.size());
        for (int i = 0; i < key.size(); i++)
        {
            items.push_back({key[i], val[i]});
            // std::cout << key[i] << ":" << val[i] << " ";
        }
        end.push_back(items.size());
        // std::cout << "\n";
    }

    int32_t *d_high_utility_patterns;
    hipMallocManaged(&d_high_utility_patterns, 128 * MEGA);
    memset(d_high_utility_patterns, 0, 128 * MEGA);
    d_high_utility_patterns[1] = 2;

    init_global_allocator(total_memory, 0);

    std::cout << "Number of Transactions: " << start.size() << "\n";
    // CudaMemoryManager *mm = createCudaMemoryManager(total_memory, page_size);
    // std::cout << "Memory Manager Initialized\n";

    AtomicWorkStack *stack;
    hipMallocManaged(&stack, sizeof(AtomicWorkStack));
    stack->init();

    WorkItem *work_item;
    hipMallocManaged(&work_item, sizeof(WorkItem));
    work_item->pattern = nullptr;
    work_item->pattern_length = 0;
    // work_item->work_count = primary.size();
    work_item->max_item = max_item;
    work_item->work_done = nullptr;

    Database *db;
    hipMallocManaged(&db, sizeof(Database));
    db->numItems = items.size();
    db->d_data = nullptr;
    db->d_transactions = nullptr;
    db->numTransactions = start.size();

    hipMallocManaged(&db->d_data, items.size() * sizeof(Item));
    hipMemcpy(db->d_data, items.data(), items.size() * sizeof(Item), hipMemcpyHostToDevice);

    hipMallocManaged(&db->d_transactions, start.size() * sizeof(Transaction));
    for (int i = 0; i < start.size(); i++)
    {
        db->d_transactions[i].data = db->d_data + start[i];
        db->d_transactions[i].utility = 0;
        db->d_transactions[i].length = end[i] - start[i];
    }

    work_item->db = db;
    work_item->max_item = max_item;

    tempWork *working;
    hipMallocManaged(&working, sizeof(tempWork));
    // hipMallocManaged(&working->db, sizeof(Database));

    hipMalloc(&working->temp_transaction, sizeof(Transaction) * db->numTransactions);
    hipMalloc(&working->local_util, sizeof(Item) * max_item * scale);
    hipMalloc(&working->hashes, sizeof(int) * db->numTransactions * scale);
    hipMalloc(&working->subtree_util, sizeof(Item) * max_item * scale);

    int *d_primary;
    hipMalloc(&d_primary, primary.size() * sizeof(int));
    hipMemcpy(d_primary, primary.data(), primary.size() * sizeof(int), hipMemcpyHostToDevice);


    // int i = 0;
    // for(auto &item : primary
    for (int i = 0; i < primary.size(); i++)
    {
        int item = primary[i];

        work_item->primary = item;
        // printf("Scanning\n");

        working->num_transactions = 0;
        working->num_items = 0;
        working->utility = 0;

        hipMemset(working->local_util, 0, sizeof(Item) * max_item * scale);
        hipMemset(working->temp_transaction, 0, sizeof(Transaction) * start.size());
        hipMemset(working->hashes, -1, sizeof(int) * db->numTransactions * scale);
        hipMemset(working->subtree_util, 0, sizeof(Item) * max_item * scale);
    

        scan<<<((work_item->db->numTransactions + threads) / threads),threads>>>(work_item, working);
        hipDeviceSynchronize();

        printf("%d:Item: %d\tUtility: %d\n", i, item, working->utility);
        if (working->utility >= args.utility)
        {
            d_high_utility_patterns[0] += 1;
            int index = d_high_utility_patterns[1];
            d_high_utility_patterns[1] += 3;
            d_high_utility_patterns[index] = item;
            d_high_utility_patterns[index + 1] = working->utility;
        }

        if (working->num_transactions == 0) continue;

    //     // printf("Copying\n");

        allocate<<<1,1>>>(working);
        hipDeviceSynchronize();

        trim_project<<<((working->num_transactions + threads) / threads),threads>>>(work_item, working, args.utility);
        hipDeviceSynchronize();

    //     // printf("Project Trim Done\n");
        finalize<<<1,32>>>(stack, work_item, working, args.utility);
        hipDeviceSynchronize();
    //     // printf("\n\n");
    //     i++;
    }

    while (stack->active > 0)
    {
        printf("Top: %d\n", stack->active);
        mine<<<blocks, threads>>>(stack, d_high_utility_patterns, args.utility);
        // print last error
        cudaErr = hipGetLastError();
        if (cudaErr != hipSuccess)
        {
            fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaErr));
            return -1;
        }

        hipDeviceSynchronize();
    }

    timer.recordPoint("Kernel Execution");
    std::map<std::string, int> Patterns = parse_patterns(d_high_utility_patterns, rename);
    // timer
    hipFree(d_high_utility_patterns);

    // for (const auto &p : Patterns)
    // {
    //     std::cout << p.first << "UTIL: " << p.second << std::endl;
    // }

    std::cout << "High Utility Patterns: " << Patterns.size() << "\n";

    // print_global_stats();

    free_global_allocator();

    timer.printRecords();

    return 0;
}