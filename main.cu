
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <thread>
#include <chrono>
#include "args.h"     // Include the args parser
#include "parser.h"   // Include the file reader
#include "work.cuh"   // Include the work queue
#include "memory.cuh" // Include the memory manager
#include "mine.cuh"

#define KILO 1024ULL
#define MEGA KILO *KILO
#define GIGA KILO *MEGA

#define page_size 512
#define total_memory 25 * GIGA

#define blocks 512
#define threads 512
// make && ./cuEFIM '/home/tarun/testing/test.txt' 5 \\s
// make && time ./cuEFIM '/home/tarun/cuEFIM/datasets/accidents_utility_spmf.txt' 15000000 \\s

std::map<std::string, int> parse_patterns(int *d_high_utility_patterns, std::unordered_map<int, std::string> rename)

{
    std::map<std::string, int> Patterns;
    int duplicate = 0;
    int util_dup = 0;

    // convert high utility patterns to string
    std::string high_utility_patterns_str = "";
    std::vector<int> high_utility_patten;
    for (int i = 0; i < d_high_utility_patterns[1]; i++)
    {
        while (d_high_utility_patterns[i + 2] != 0)
        {
            high_utility_patten.push_back(d_high_utility_patterns[i + 2]);
            i++;
        }

        // if empty, skip
        if (high_utility_patten.size() == 0)
        {
            continue;
        }

        for (int j = 0; j < high_utility_patten.size() - 1; j++)
        {
            high_utility_patterns_str += rename[high_utility_patten[j]] + " ";
        }

        //

        // Patterns[high_utility_patterns_str] = high_utility_patten[high_utility_patten.size() - 1];
        if (Patterns.find(high_utility_patterns_str) == Patterns.end())
        {
            Patterns[high_utility_patterns_str] = high_utility_patten[high_utility_patten.size() - 1];
        }
        else
        {
            duplicate++;
            // print old util and new util
            // std::cout << "Old: " << Patterns[high_utility_patterns_str] << " New: " << high_utility_patten[high_utility_patten.size() - 1] << "\n";
            if (Patterns[high_utility_patterns_str] == high_utility_patten[high_utility_patten.size() - 1])
            {
                util_dup++;
            }
        }

        high_utility_patterns_str = "";
        high_utility_patten.clear();
    }
    std::cout << "Duplicate: " << duplicate << "\n";
    std::cout << "Util Duplicate: " << util_dup << "\n";
    return Patterns;
}

int main(int argc, char *argv[])
{
    // Make CPU not poll
    hipError_t err = hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to set device flags: %s\n", hipGetErrorString(err));
        return -1;
    }

    // increase stack size
    // hipDeviceSetLimit(hipLimitStackSize, 64 * 1024);

    // Parse command-line arguments using args_parser
    ParsedArgs args;
    if (!parseArguments(argc, argv, args))
    {
        // Parsing failed; exit the program
        return EXIT_FAILURE;
    }

    // increase cuda stack size
    // hipDeviceSetLimit(hipLimitStackSize, 32 * 1024);

    ReadFileResult fileResult = read_file(args.filename, args.separator, args.utility);

    // Access the parsed data
    auto &filteredTransactions = fileResult.filteredTransactions;
    auto &primary = fileResult.primary;
    auto &rename = fileResult.rename;
    int max_item = fileResult.max_item;

    // Flatten filteredTransactions
    std::vector<Item> items;
    std::vector<int> start;
    std::vector<int> end;

    for (const auto &[key, val] : filteredTransactions)
    {
        start.push_back(items.size());
        for (int i = 0; i < key.size(); i++)
        {
            // std::cout << key[i] << ":" << val[i] << " ";
            items.push_back({key[i], val[i]});
        }
        // std::cout << "\n";
        end.push_back(items.size());
    }
    // std::cout << "\n";

    int32_t *d_high_utility_patterns;
    hipMallocManaged(&d_high_utility_patterns, 128 * MEGA);
    memset(d_high_utility_patterns, 0, 128 * MEGA);
    d_high_utility_patterns[1] = 2;

    // using MemoryManagerType = OuroVAPQ;
    // MemoryManagerType mm;
    // mm.initialize(0, 1024 * 1024 * 1024);

    CudaMemoryManager *mm = createCudaMemoryManager(total_memory / page_size, page_size);
    AtomicWorkStack *curr_work_queue;
    hipMallocManaged(&curr_work_queue, sizeof(AtomicWorkStack));
    curr_work_queue->init();

    Item *d_items = reinterpret_cast<Item *>(mm->host_malloc(items.size() * sizeof(Item)));
    int *d_start = reinterpret_cast<int *>(mm->host_malloc(start.size() * sizeof(int)));
    int *d_end = reinterpret_cast<int *>(mm->host_malloc(end.size() * sizeof(int)));
    int *d_primary = reinterpret_cast<int *>(mm->host_malloc(primary.size() * sizeof(int)));

    // copy items to device
    hipMemcpy(d_items, items.data(), items.size() * sizeof(Item), hipMemcpyHostToDevice);
    hipMemcpy(d_start, start.data(), start.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_end, end.data(), end.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_primary, primary.data(), primary.size() * sizeof(int), hipMemcpyHostToDevice);

    WorkItem work_item;
    work_item.pattern = (int *)mm->host_malloc(sizeof(int));
    work_item.pattern_length = 0;

    work_item.db = (Database *)mm->host_malloc(sizeof(Database));
    work_item.db->numItems = items.size();

    work_item.db->d_data = (Item *)mm->host_malloc(items.size() * sizeof(Item));
    memcpy(work_item.db->d_data, d_items, items.size() * sizeof(Item));

    // work_item.db->d_transactions = reinterpret_cast<Transaction *>(mm.hostMalloc(start.size() * sizeof(Transaction)));
    work_item.db->d_transactions = (Transaction *)mm->host_malloc(start.size() * sizeof(Transaction));
    work_item.db->numTransactions = start.size();
    for (int i = 0; i < start.size(); i++)
    {
        work_item.db->d_transactions[i].data = work_item.db->d_data + d_start[i];
        work_item.db->d_transactions[i].utility = 0;
        work_item.db->d_transactions[i].length = d_end[i] - d_start[i];
    }

    work_item.db->numItems = items.size();

    // work_item.work_done = reinterpret_cast<int *>(mm.hostMalloc(sizeof(int)));
    work_item.work_done = (int *)mm->host_malloc(sizeof(int));
    work_item.work_count = primary.size();
    work_item.max_item = max_item;

    for (int i = 0; i < primary.size(); i++)
    {
        work_item.primary = d_primary[i];
        curr_work_queue->host_push(work_item);
    }


    // hipError_t cudaStatus = hipGetLastError();
    auto starttime = std::chrono::high_resolution_clock::now();
    hipError_t cudaStatus;

    while(curr_work_queue->active > 0)
    {
        printf("Top: %d\n", curr_work_queue->active);
        test<<<blocks, threads>>>(curr_work_queue, d_high_utility_patterns, mm, args.utility);
        // print last error
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return -1;
        }

        hipDeviceSynchronize();
    }

    auto endtime = std::chrono::high_resolution_clock::now();

    std::cout << "GPU time: " << std::chrono::duration_cast<std::chrono::milliseconds>(endtime - starttime).count() / 1000.0 << " s\n";


    // std::cout << "High Utility Patterns: " << d_high_utility_patterns[0] << "\n";
    std::map<std::string, int> Patterns = parse_patterns(d_high_utility_patterns, rename);
    hipFree(d_high_utility_patterns);

    // for (const auto &p : Patterns)
    // {
    //     std::cout << p.first << "UTIL: " << p.second << std::endl;
    // }

    std::cout << "High Utility Patterns: " << Patterns.size() << "\n";

    return 0;
}