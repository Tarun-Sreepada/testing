#include "hip/hip_runtime.h"

#include "allocator.cuh"

#define KILO 1024ULL
#define MEGA KILO *KILO
#define GIGA KILO *MEGA

#define BUCKET_SCALE 3
#define WORK_QUEUE_CAPACITY 8192

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

#include <assert.h>
#define cdpErrchk(ans)                        \
    {                                         \
        cdpAssert((ans), __FILE__, __LINE__); \
    }
__device__ void cdpAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        printf("GPU kernel assert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            assert(0);
    }
}

__device__ uint32_t pcg_hash(uint32_t input)
{
    uint32_t state = input * 747796405u + 2891336453u;
    uint32_t word = ((state >> ((state >> 28u) + 4u)) ^ state) * 277803737u;
    return (word >> 22u) ^ word;
}

__device__ uint32_t hashFunction(uint32_t key, uint32_t tableSize)
{
    return pcg_hash(key) % tableSize;
}

__device__ uint32_t array_hasher(Item *array, uint32_t size)
{
    uint32_t hash = 0;
    for (uint32_t i = 0; i < size; i++)
    {
        hash = pcg_hash(hash + array[i].id);
    }
    return hash;
}

__device__ uint32_t array_hasher_function(Item *array, uint32_t size, uint32_t tableSize)
{
    uint32_t hash = array_hasher(array, size);
    return hash % tableSize;
}

__device__ int binarySearchItems(const Item *items, int n, uint32_t search_id, int offset, int length)
{
    // Validate that the provided range is within bounds.
    if (offset < 0 || offset >= n || length <= 0 || (offset + length) > n)
    {
        return -1;
    }

    int l = offset;
    int r = offset + length - 1;
    while (l <= r)
    {
        int mid = l + (r - l) / 2;
        uint32_t mid_id = items[mid].id;
        if (mid_id == search_id)
        {
            return mid;
        }
        else if (mid_id < search_id)
        {
            l = mid + 1;
        }
        else
        {
            r = mid - 1;
        }
    }
    return -1;
}

// Helper: split a string by a delimiter.
std::vector<std::string> split(const std::string &str, const std::string &delimiter)
{
    std::vector<std::string> tokens;
    size_t prev = 0, pos = 0;
    while ((pos = str.find(delimiter, prev)) != std::string::npos)
    {
        tokens.push_back(str.substr(prev, pos - prev));
        prev = pos + delimiter.length();
    }
    tokens.push_back(str.substr(prev));
    return tokens;
}


__global__ void mine_kernel_buffer(PageAllocator *alloc, AtomicWorkQueue<WorkItem, WORK_QUEUE_CAPACITY> *workQueue, uint32_t minUtil, uint32_t *high_utility_patterns)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // get the workitem from the work queue and print number of primary
    WorkItem wi;
    while (workQueue->get_work_count() > 0)
    {
        if (!workQueue->dequeue(wi)) {
            // printf("Work Count: %d\n", workQueue->get_work_count());
            continue;
        }


        uint32_t scratch_bytes = (wi.pattern[0] + 1) * sizeof(uint32_t) +
                                (sizeof(uint32_t) * 2) + // work_done, work_count
                                 sizeof(Item) * wi.num_items +
                                 sizeof(uint32_t) * wi.num_transactions * 3 + // start, end, utilities
                                 sizeof(Item) * wi.max_item * BUCKET_SCALE * 2; // local_util, subtree_util
        
        void *scratch = page_alloc_device(alloc, scratch_bytes);
        void *base_ptr = scratch;

        memset(scratch, 0, scratch_bytes);



        uint32_t *n_pattern = (uint32_t *)scratch;
        scratch += (wi.pattern[0] + 1) * sizeof(uint32_t);
        Item *scratch_items = (Item *)scratch;
        scratch += sizeof(Item) * wi.num_items;
        uint32_t *scratch_start = (uint32_t *)scratch;
        scratch += sizeof(uint32_t) * wi.num_transactions;
        uint32_t *scratch_end = (uint32_t *)scratch;
        scratch += sizeof(uint32_t) * wi.num_transactions;
        uint32_t *scratch_utility = (uint32_t *)scratch;
        scratch += sizeof(uint32_t) * wi.num_transactions;
        Item *local_util = (Item *)scratch;
        scratch += sizeof(Item) * wi.max_item * BUCKET_SCALE;
        Item *subtree_util = (Item *)scratch;
        scratch += sizeof(Item) * wi.max_item * BUCKET_SCALE;
        uint32_t *work_done = (uint32_t *)scratch;
        scratch += sizeof(uint32_t);
        uint32_t *work_count = (uint32_t *)scratch;



        uint32_t item_counter = 0;
        uint32_t transaction_counter = 0;
        uint32_t pattern_utility = 0;

        // printf("Primary Item: %d\n", wi.primary_item);

        for (int j = 0; j < wi.num_transactions; j++)
        {
            //  if start == end continue
            if (wi.start[j] == wi.end[j])
            {
                continue;
            }
            int idx = binarySearchItems(wi.items, wi.num_items, wi.primary_item, wi.start[j], wi.end[j] - wi.start[j]);
            uint32_t temp_util = 0;

            if (idx != -1)
            {
                scratch_start[transaction_counter] = item_counter;
                scratch_utility[transaction_counter] = wi.items[idx].utility + wi.utility[j];
                pattern_utility += scratch_utility[transaction_counter];
                temp_util = scratch_utility[transaction_counter];

                for (int k = idx + 1; k < wi.end[j]; k++)
                {
                    // printf("%d:%d ", wi.items[k].id, wi.items[k].utility);
                    scratch_items[item_counter] = wi.items[k];
                    temp_util += wi.items[k].utility;
                    item_counter++;
                }
                // printf("|| %d ||%d\n", scratch_utility[transaction_counter], temp_util);

                if (item_counter != scratch_start[transaction_counter])
                {
                    scratch_end[transaction_counter] = item_counter;
                    transaction_counter++;
                }

                for (int k = idx + 1; k < wi.end[j]; k++)
                {
                    // printf("Item: %d\n", wi.items[k].id);
                    // we do a hash of the item id to find where to put it in the local utility
                    uint32_t hash = hashFunction(wi.items[k].id, wi.max_item * BUCKET_SCALE);
                    while (true)
                    {
                        // printf("Hash: %d\tlocal_util[hash].id: %d\n", hash, local_util[hash].id);

                        if (local_util[hash].id == 0 || local_util[hash].id == wi.items[k].id)
                        {
                            local_util[hash].id = wi.items[k].id;
                            local_util[hash].utility += temp_util;
                            break;
                        }
                        else
                        {
                            // linear probing
                            hash = (hash + 1) % (wi.max_item * BUCKET_SCALE);
                        }
                    }
                }
            }
        }


        // allocate new pattern if utility is greater than minUtil or transaction counter is greater than 1
        if (pattern_utility >= minUtil || transaction_counter)
        {
            // n_pattern = (uint32_t *)bump_alloc(alloc, sizeof(uint32_t) * (wi.pattern[0] + 3));
            n_pattern = (uint32_t *)page_alloc_device(alloc, sizeof(uint32_t) * (wi.pattern[0] + 3));
            if (!n_pattern)
            {
                printf("Error allocating new pattern.\n");
                return;
            }

            memcpy(n_pattern, wi.pattern, sizeof(uint32_t) * (wi.pattern[0] + 1));
            n_pattern[0] += 1;
            n_pattern[n_pattern[0]] = wi.primary_item;

            // print pattern
            // printf("Pattern: ");
            // for (int j = 0; j < n_pattern[0]; j++)
            // {
            //     printf("%d ", n_pattern[j + 1]);
            // }
            // printf(": %d\n", pattern_utility);
        }

        if (pattern_utility >= minUtil)
        {
            // add pattern to high utility patterns [0 is number of patterns][1 is offset to start wiritn from]
            uint32_t ret = atomicAdd(&high_utility_patterns[0], 1);
            printf("Pat count: %d\tTID: %d\n", ret, tid);
            uint32_t offset = atomicAdd(&high_utility_patterns[1], n_pattern[0] + 2); // 1 for utilty and 1 spacer
            for (int j = 0; j < n_pattern[0]; j++)
            {
                high_utility_patterns[offset + j] = n_pattern[j + 1];
            }
            high_utility_patterns[offset + n_pattern[0]] = pattern_utility;

        }

        if (transaction_counter)
        {
            // count number of local utility that is greater than minUtil
            uint32_t local_util_counter = 0;
            for (int j = 0; j < wi.max_item * BUCKET_SCALE; j++)
            {
                if (local_util[j].utility >= minUtil)
                {
                    local_util_counter++;
                }
            }

            uint32_t new_item_count = 0;
            uint32_t new_transaction_count = 0;


            for (int j = 0; j < wi.num_transactions; j++)
            {
                if (scratch_start[j] != scratch_end[j])
                {
                    uint32_t start = new_item_count;

                    for (int k = scratch_start[j]; k < scratch_end[j]; k++)
                    {
                        // check if the item is in the local utility and if it is greater than minUtil
                        uint32_t hash = hashFunction(scratch_items[k].id, wi.max_item * BUCKET_SCALE);
                        while (true)
                        {
                            if (local_util[hash].id == scratch_items[k].id)
                            {
                                if (local_util[hash].utility >= minUtil)
                                {
                                    scratch_items[new_item_count] = scratch_items[k];
                                    new_item_count++;
                                }
                                break;
                            }
                            else
                            {
                                hash = (hash + 1) % (wi.max_item * BUCKET_SCALE);
                            }
                        }
                    }
                
                    if (new_item_count != start)
                    {
                        scratch_start[new_transaction_count] = start;
                        scratch_end[new_transaction_count] = new_item_count;
                        scratch_utility[new_transaction_count] = scratch_utility[j];
                        new_transaction_count++;
                    }

                    if (new_transaction_count == transaction_counter)
                    {
                        break;
                    }

                }
            }

            for (int j = 0; j < new_transaction_count; j++)
            {
                // printf("%d|" , scratch_utility[j]);
                // for (int k = scratch_start[j]; k < scratch_end[j]; k++)
                // {
                //     printf("%d:%d ", scratch_items[k].id, scratch_items[k].utility);
                // }
                // printf("\n");

                uint32_t temp_util = scratch_utility[j];
                for (int k = scratch_start[j]; k < scratch_end[j]; k++)
                {
                    temp_util += scratch_items[k].utility;
                }
                uint32_t temp = 0;
                // printf("%d|", scratch_utility[j]);

                for (int k = scratch_start[j]; k < scratch_end[j]; k++)
                {
                    uint32_t hash = hashFunction(scratch_items[k].id, wi.max_item * BUCKET_SCALE);
                    // printf("%d:%d ", scratch_items[k].id, scratch_items[k].utility);
                    while (true)
                    {
                        if (subtree_util[hash].id == scratch_items[k].id || subtree_util[hash].id == 0)
                        {
                            subtree_util[hash].id = scratch_items[k].id;
                            subtree_util[hash].utility += temp_util - temp;
                            temp += scratch_items[k].utility;
                            break;
                        }
                        else
                        {
                            hash = (hash + 1) % (wi.max_item * BUCKET_SCALE);
                        }
                    }
                }
                // printf("\n");

            }

            uint32_t primary_count = 0;
            // printf("Subtree Util: ");
            for (int j = 0; j < wi.max_item * BUCKET_SCALE; j++)
            {
                if (subtree_util[j].id == 0)
                    continue;
                // printf("%d:%d ", subtree_util[j].id, subtree_util[j].utility);
                if (subtree_util[j].utility >= minUtil)
                {
                    primary_count++;
                }
            }
            // printf("\n");

            // printf("Primary Count: %d\n", primary_count);



            if (primary_count)
            {
                // create new work item for each primary
                for (int j = 0; j < wi.max_item * BUCKET_SCALE; j++)
                {
                    if (subtree_util[j].utility >= minUtil)
                    {
                        WorkItem this_thing;

                        this_thing.pattern = n_pattern;
                        this_thing.items = scratch_items;
                        this_thing.num_items = new_item_count;
                        this_thing.start = scratch_start;
                        this_thing.end = scratch_end;
                        this_thing.utility = scratch_utility;
                        this_thing.num_transactions = new_transaction_count;
                        this_thing.primary_item = subtree_util[j].id;
                        this_thing.max_item = local_util_counter;
                        this_thing.base_ptr = base_ptr;
                        this_thing.bytes = scratch_bytes;
                        this_thing.work_done = work_done;
                        this_thing.work_count = primary_count;

                        while (!workQueue->enqueue(this_thing))
                        {
                            printf("Waiting for space in the queue.\n");
                        }
                    }
                }
            }

            else
            {
                // free memory
                page_free_device(alloc, base_ptr, scratch_bytes);
            }

        }
        else
        {
            // free memory
            page_free_device(alloc, base_ptr, scratch_bytes);
        }
        uint32_t done = atomicAdd(wi.work_done, 1);
        uint32_t old = atomicSub(&workQueue->work_count, 1);
        printf("tid: %d\tDone: %d\tLocal Work Count: %d\tGlobal Work Count: %d\n", tid, done, wi.work_count, workQueue->work_count);
        // if old == work_done - 1, free the memory

        if (done >= wi.work_count - 1)
        {
            // printf("Freeing memory\n");
            page_free_device(alloc, wi.base_ptr, wi.bytes);
        }
        // printf("\n");

    }
}

class cuEFIM
{
public:
    cuEFIM(const std::string &iFile, int minUtil, const std::string &sep = "\t", uint64_t alloc_size = 4ULL * GIGA) : inputFile(iFile), minUtil(minUtil), runtime(0), alloc_size(alloc_size)
    {
        if (sep == "\\s")
            this->sep = " ";
        else if (sep == "\\t")
            this->sep = "\t";
        else
            this->sep = sep;
    }

    void mine()
    {



        auto start_time = std::chrono::high_resolution_clock::now();

        auto [transactions, primary] = _read_file();

        std::cout << "Time to read file: " << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - start_time).count() << " ms" << std::endl;

        std::cout << "Number of transactions: " << transactions.size() << std::endl;
        std::cout << "Number of items: " << rename.size() << std::endl;
        std::cout << "Max item id: " << max_item << std::endl;

        // // print all transactions
        // std::cout << "Transactions: " << std::endl;
        // for (const auto &t : transactions)
        // {
        //     for (int i = 0; i < t.first.size(); i++)
        //     {
        //         std::cout << t.first[i] << ":" << t.second[i] << " ";
        //     }
        //     std::cout << std::endl;
        // }

        // // print all primary
        // std::cout << "Primary: " << std::endl;
        // for (const auto &p : primary)
        // {
        //     std::cout << p << " ";
        // }
        // std::cout << std::endl;
        // std::cout << std::endl;


        // prepare indices for device memory
        std::vector<uint32_t> start;
        std::vector<uint32_t> end;
        std::vector<Item> items;
        std::vector<uint32_t> util(transactions.size(), 0);

        uint32_t counter = 0;
        for (const auto &t : transactions)
        {
            start.push_back(counter);
            for (size_t i = 0; i < t.first.size(); ++i)
            {
                items.push_back(Item{static_cast<uint32_t>(t.first[i]), static_cast<uint32_t>(t.second[i])});
                counter++;
            }
            end.push_back(counter);
        }

        // create BumpAllocator
        std::cout << "Allocating " << alloc_size << " bytes for the bump allocator." << std::endl;

        PageAllocator *page_alloc = createUnifiedPageAllocator(alloc_size);
        if (!page_alloc)
        {
            std::cerr << "Error creating page allocator." << std::endl;
            return;
        }

        // BumpAllocator *alloc = createUnifiedBumpAllocator(alloc_size);
        // if (!alloc)
        // {
        //     std::cerr << "Error creating bump allocator." << std::endl;
        //     return;
        // }

        // // print alloc bytes used
        // double mb_used = static_cast<double>(alloc->offset) / static_cast<double>(MEGA);

        uint32_t bytes_initial_acquired = 1 * sizeof(uint32_t) + // pattern
                                            1 * sizeof(uint32_t) + // work done count
                                            items.size() * sizeof(Item) +
                                            start.size() * sizeof(uint32_t) +
                                            end.size() * sizeof(uint32_t) +
                                            util.size() * sizeof(uint32_t) +
                                            primary.size() * sizeof(uint32_t);

        void *initial_acquired = page_alloc_host(page_alloc, bytes_initial_acquired);
        if (!initial_acquired)
        {
            std::cerr << "Error allocating initial memory." << std::endl;
            return;
        }

        // Allocate memory for the pattern, items, start, end, utility, and primary.
        uint32_t *d_pattern = (uint32_t *)initial_acquired; d_pattern[0] = 0;
        uint32_t *d_work_done = d_pattern + sizeof(uint32_t); d_work_done[0] = 0;
        Item *d_items = (Item *)(d_work_done + sizeof(uint32_t));

        uint32_t *d_start = (uint32_t *)(d_items + items.size() * sizeof(Item));
        uint32_t *d_end = (uint32_t *)(d_start + start.size() * sizeof(uint32_t));
        uint32_t *d_utility = (uint32_t *)(d_end + end.size() * sizeof(uint32_t));
        uint32_t *d_primary = (uint32_t *)(d_utility + util.size() * sizeof(uint32_t));

        // uint32_t pattern_init = 0;
        // uint32_t *d_pattern = bump_allocate_and_copy(alloc, &pattern_init, 1);
        // Item *d_items = bump_allocate_and_copy(alloc, items.data(), items.size());
        // uint32_t *d_start = bump_allocate_and_copy(alloc, start.data(), start.size());
        // uint32_t *d_end = bump_allocate_and_copy(alloc, end.data(), end.size());
        // uint32_t *d_utility = bump_allocate_and_copy(alloc, util.data(), util.size());
        // uint32_t *d_primary = bump_allocate_and_copy(alloc, primary.data(), primary.size());

        // mb_used = static_cast<double>(alloc->offset) / static_cast<double>(MEGA);




        // // Copy the data to device memory.
        hipMemcpy(d_start, start.data(), start.size() * sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(d_end, end.data(), end.size() * sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(d_primary, primary.data(), primary.size() * sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(d_items, items.data(), items.size() * sizeof(Item), hipMemcpyHostToDevice);
        hipMemset(d_utility, 0, start.size() * sizeof(uint32_t));



        uint32_t *d_high_utility_patterns;
        hipMallocManaged(&d_high_utility_patterns, MEGA);
        d_high_utility_patterns[1] = 2;

        std::cout << "Time to allocate and copy data to device: " << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - start_time).count() << " ms" << std::endl;

        AtomicWorkQueue<WorkItem, WORK_QUEUE_CAPACITY> *workQueue;
        hipMallocManaged(&workQueue, sizeof(AtomicWorkQueue<WorkItem, WORK_QUEUE_CAPACITY>));
        workQueue->init();

        for (const auto &p : primary)
        {
            WorkItem initial;
            initial.base_ptr = initial_acquired;
            initial.bytes = bytes_initial_acquired;
            initial.work_done = d_work_done;
            initial.work_count = primary.size();

            initial.pattern = d_pattern;
            initial.items = d_items;
            initial.num_items = items.size();
            initial.start = d_start;
            initial.end = d_end;
            initial.utility = d_utility;
            initial.num_transactions = transactions.size();
            initial.primary_item = p;
            initial.max_item = max_item;


            workQueue->host_enqueue(initial);
        }


        // print work count
        printf("Work Count: %d\n", workQueue->work_count);

        mine_kernel_buffer<<<16, 1>>>(page_alloc, workQueue, minUtil, d_high_utility_patterns);
        hipDeviceSynchronize();
        std::cout << "Work Count: " << workQueue->work_count << std::endl;

        // while (workQueue->work_count > 0)
        // {
        //     mine_kernel_buffer<<<64, 1>>>(alloc, workQueue, minUtil, d_high_utility_patterns);
        //     hipDeviceSynchronize();
        //     std::cout << "Work Count: " << workQueue->work_count << std::endl;
        //     std::cout << "Patterns: " << d_high_utility_patterns[0] << std::endl;   
        //     // print pattern of WorkItem at the top of the queue
            
        // }

        runtime = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - start_time).count();

        freeUnifiedPageAllocator(page_alloc);
        printf("Number of high utility patterns: %d\n", d_high_utility_patterns[0]);

        // convert high utility patterns to string
        std::string high_utility_patterns_str = "";
        std::vector<int> high_utility_patten;
        for (int i = 0; i < d_high_utility_patterns[1]; i++)
        {
            while (d_high_utility_patterns[i + 2] != 0)
            {
                high_utility_patten.push_back(d_high_utility_patterns[i + 2]);
                i++;
            }
            // if empty, skip
            if (high_utility_patten.size() == 0)
            {
                continue;
            }

            for (int j = 0; j < high_utility_patten.size() - 1; j++)
            {
                high_utility_patterns_str += std::to_string(high_utility_patten[j]) + " ";
            }
            Patterns[high_utility_patterns_str] = high_utility_patten[high_utility_patten.size() - 1];

            high_utility_patterns_str = "";
            high_utility_patten.clear();
        }
        // std::cout << high_utility_patterns_str << std::endl;
        for (const auto &p : Patterns)
        {
            std::cout << p.first << "UTIL: " << p.second << std::endl;
        }
    }

    // Print final results.
    void printResults() const
    {
        std::cout << "Total number of High Utility Patterns: " << Patterns.size() << "\n";
        std::cout << "Total Execution Time (seconds): " << runtime / 1000 << "\n";
        // Memory info is platform‐dependent in C++ so is omitted here.
    }

    // Get discovered patterns.
    const std::map<std::string, int> &getPatterns() const
    {
        return Patterns;
    }

    double getRuntime() const { return runtime; }

    // Save results to a file.
    void save(const std::string &outFile)
    {
        std::ofstream writer(outFile);
        if (!writer)
        {
            std::cerr << "Error opening output file: " << outFile << "\n";
            return;
        }
        // Here we write the discovered patterns (stored in Patterns)
        for (const auto &entry : Patterns)
        {
            writer << entry.first << ":" << entry.second << "\n";
        }
        writer.close();
    }

private:
    std::string inputFile;
    std::string sep;
    uint32_t minUtil;
    uint64_t alloc_size;

    uint32_t max_item;

    std::unordered_map<uint32_t, std::string> rename;

    std::map<std::string, int> Patterns;

    double runtime;

    void _increase_cuda_stack()
    {
        size_t currentStackSize;
        hipDeviceGetLimit(&currentStackSize, hipLimitStackSize);
        printf("Current device stack size: %zu bytes\n", currentStackSize);

        // hipDeviceSetLimit(hipLimitStackSize, 8192);
        size_t stackSize = 64 * KILO; // 8192 per thread (adjust as needed)
        hipError_t err = hipDeviceSetLimit(hipLimitStackSize, stackSize);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to set device stack size (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        hipDeviceGetLimit(&currentStackSize, hipLimitStackSize);
        printf("Current device stack size: %zu bytes\n", currentStackSize);
    }

    std::pair<std::map<std::vector<int>, std::vector<int>>, std::vector<uint32_t>> _read_file()
    {
        // File data: each entry is a pair: (vector of item strings, vector of utility ints)
        std::vector<std::pair<std::vector<std::string>, std::vector<int>>> fileData;
        // TWU dictionary: item string -> total weight
        std::unordered_map<std::string, int> twu;

        std::ifstream infile(inputFile);
        if (!infile)
        {
            std::cerr << "Error opening file: " << inputFile << "\n";
            exit(1);
        }
        std::string line;
        while (std::getline(infile, line))
        {
            // Expected format: items : weight : utility_list
            // Items and utility_list are further separated by sep.
            std::vector<std::string> parts = split(line, ":");
            if (parts.size() < 3)
                // continue;
                std::cerr << "Invalid line: " << line << "\n";

            std::vector<std::string> items = split(parts[0], sep);

            int weight = std::stoi(parts[1]);
            std::vector<std::string> utilStrs = split(parts[2], sep);

            std::vector<int> utils;
            for (const auto &s : utilStrs)
                utils.push_back(std::stoi(s));
            fileData.push_back({items, utils});
            // Update TWU: add weight for each item
            for (const auto &item : items)
            {
                twu[item] += weight;
            }
        }
        infile.close();

        // // print all twu
        // std::cout << "TWU: " << std::endl;
        // for (const auto &t : twu) {
        //     std::cout << t.first << ":" << t.second << std::endl;
        // }

        // Filter twu based on minUtil threshold.
        for (auto it = twu.begin(); it != twu.end();)
        {
            if (it->second < minUtil)
                it = twu.erase(it);
            else
                ++it;
        }

        // Create a sorted vector of (item, utility) in descending order by utility.
        std::vector<std::pair<std::string, int>> sortedTWU(twu.begin(), twu.end());
        std::sort(sortedTWU.begin(), sortedTWU.end(),
                  [](const auto &a, const auto &b)
                  { return a.second > b.second; });

        // Map each item (string) to an integer (starting from count downwards).
        std::unordered_map<std::string, int> strToInt;
        int t = static_cast<int>(sortedTWU.size());
        max_item = t;
        for (const auto &p : sortedTWU)
        {
            strToInt[p.first] = t;
            rename[t] = p.first;
            t--;
        }

        // Build filtered transactions and compute subtree utility.
        std::unordered_map<int, int> subtree;
        std::map<std::vector<int>, std::vector<int>> filteredTransactions;
        for (const auto &entry : fileData)
        {
            const std::vector<std::string> &items = entry.first;
            const std::vector<int> &utils = entry.second;
            std::vector<std::pair<int, int>> transaction;
            for (size_t i = 0; i < items.size(); ++i)
            {
                if (strToInt.find(items[i]) != strToInt.end())
                {
                    transaction.push_back({strToInt[items[i]], utils[i]});
                }
            }
            if (transaction.empty())
                continue;
            // Sort transaction by item id.
            std::sort(transaction.begin(), transaction.end(),
                      [](const std::pair<int, int> &a, const std::pair<int, int> &b)
                      { return a.first < b.first; });

            std::vector<int> key;
            std::vector<int> val;
            for (const auto &p : transaction)
            {
                key.push_back(p.first);
                val.push_back(p.second);
            }

            // Use key (sorted vector) as key for filteredTransactions.
            if (filteredTransactions.find(key) == filteredTransactions.end())
            {
                // filteredTransactions[key] = Transaction{key, val, 0};
                filteredTransactions[key] = val;
            }
            else
            {
                // If the transaction already exists, add corresponding utilities.
                // Transaction &trans = filteredTransactions[key];
                for (size_t i = 0; i < val.size(); ++i)
                    // trans.vals[i] += val[i];
                    filteredTransactions[key][i] += val[i];
            }

            // Compute subtree utility for this transaction.
            int subUtil = 0;
            for (int v : val)
                subUtil += v;
            int temp = 0;
            for (size_t i = 0; i < key.size(); ++i)
            {
                subtree[key[i]] += subUtil - temp;
                temp += val[i];
            }
        }

        // Determine primary items: those with subtree utility >= minUtil.
        std::vector<uint32_t> primary;
        for (const auto &p : subtree)
        {
            if (p.second >= minUtil)
                primary.push_back(p.first);
        }

        return std::make_pair(filteredTransactions, primary);
    }
};

int main(int argc, char *argv[])
{
    if (argc < 4 || argc > 6)
    {
        std::cerr << "Usage: " << argv[0] << " <filename> <delimiter> <minUtil> [<alloc_size>] [<output_file>]" << std::endl;
        return 1;
    }

    std::string filename(argv[1]);
    std::string delimiter(argv[2]);
    uint32_t minUtil;

    try
    {
        minUtil = std::stoi(argv[3]);
    }
    catch (const std::exception &e)
    {
        std::cerr << "Error parsing command line arguments: " << e.what() << std::endl;
        return 1;
    }

    if (argc == 5)
    {
        cuEFIM efim(filename, minUtil, delimiter, std::stoull(argv[4]) * GIGA);
        efim.mine();
        efim.printResults();
    }
    else if (argc == 6)
    {
        cuEFIM efim(filename, minUtil, delimiter, std::stoull(argv[4]) * GIGA);
        efim.mine();
        efim.printResults();
        efim.save(argv[5]);
    }
    else
    {
        cuEFIM efim(filename, minUtil, delimiter);
        efim.mine();
        efim.printResults();
    }

    return 0;
}
